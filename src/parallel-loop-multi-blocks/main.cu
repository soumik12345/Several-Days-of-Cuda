#include "hip/hip_runtime.h"


#include <stdio.h>


__global__ void parallel_for_loop() {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Current Iteration Number: %d\n", index);
}


class ParallelizedForLoopProgramMultipleBlocks {

public:
    int nBlocks, nThreads;

    ParallelizedForLoopProgramMultipleBlocks(int nBlocks, int nThreads);
    void run();
};

ParallelizedForLoopProgramMultipleBlocks::ParallelizedForLoopProgramMultipleBlocks(int nBlocks, int nThreads) {
    this->nBlocks = nBlocks;
    this->nThreads = nThreads;
}

void ParallelizedForLoopProgramMultipleBlocks::run() {

    int deviceId = hipGetDevice(&deviceId);

    printf("GPU Device ID: %d\n", deviceId);
    printf("CPU Device ID: %d\n\n", hipCpuDeviceId);

    parallel_for_loop<<<this->nBlocks, this->nThreads>>>();
    hipDeviceSynchronize();
}

int main() {

    ParallelizedForLoopProgramMultipleBlocks program(10, 1);
    program.run();
}
