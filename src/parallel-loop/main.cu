#include "hip/hip_runtime.h"


#include <stdio.h>


__global__ void parallel_for_loop() {
    printf("Current Iteration Number: %d\n", threadIdx.x);
}


class ParallelizedForLoopProgram {

public:
    int n;

    ParallelizedForLoopProgram(int n);
    void run();
};

ParallelizedForLoopProgram::ParallelizedForLoopProgram(int n) {
    this->n = n;
}

void ParallelizedForLoopProgram::run() {

    int deviceId = hipGetDevice(&deviceId);

    printf("GPU Device ID: %d\n", deviceId);
    printf("CPU Device ID: %d\n\n", hipCpuDeviceId);

    parallel_for_loop<<<1, this->n>>>();
    hipDeviceSynchronize();
}

int main() {

    ParallelizedForLoopProgram program(10);
    program.run();
}
