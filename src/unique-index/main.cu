#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

struct BlockParams {
    int x, y, z;
};


struct GridParams {
    int x, y, z;
};

struct ThreadParams {
    int x, y, z;
};


__global__ void unique_index_calculation_kernel(int* input) {

    int threadId = threadIdx.x;
    int offset = blockIdx.x * blockDim.x;
    int globalIndex = threadId + offset;
    printf(
            "threadIdx.x = %d, offset = %d, globalIndex = %d, Value = %d\n",
            threadId, offset, globalIndex, input[globalIndex]
    );
}


class UniqueIndexCalculation {

public:

    UniqueIndexCalculation(BlockParams, GridParams);
    UniqueIndexCalculation(BlockParams, ThreadParams);

    ~UniqueIndexCalculation();

    void run(int*, int, int);

    BlockParams BlockParameters;
    GridParams GridParameters;
};


UniqueIndexCalculation::UniqueIndexCalculation(BlockParams bParams, GridParams gParams) {

    BlockParameters = bParams;
    GridParameters = gParams;
}


UniqueIndexCalculation::UniqueIndexCalculation(BlockParams bParams, ThreadParams tParams) {

    BlockParameters = bParams;
    GridParameters = {
            tParams.x / bParams.x,
            tParams.y / bParams.y,
            tParams.z / bParams.z,
    };
}


UniqueIndexCalculation::~UniqueIndexCalculation() {

    hipDeviceReset();
}


void UniqueIndexCalculation::run(int* inputArray, int arraySize, int arraySizeBytes) {

    int deviceId = hipGetDevice(&deviceId);

    printf("GPU Device ID: %d\n", deviceId);
    printf("CPU Device ID: %d\n\n", hipCpuDeviceId);

    printf("Array Data: ");
    for(int i = 0; i < arraySize; i++)
        printf("%d ", inputArray[i]);
    printf("\n\n");

    int* gpuData;
    hipMalloc((void**)&gpuData, arraySizeBytes);
    hipMemcpy(gpuData, inputArray, arraySizeBytes, hipMemcpyHostToDevice);

    dim3 block(BlockParameters.x, BlockParameters.y, BlockParameters.z);
    dim3 grid(GridParameters.x, GridParameters.y, GridParameters.z);

    unique_index_calculation_kernel << <grid, block >> > (gpuData);
    hipDeviceSynchronize();
}


inline void Demo() {

    int arraySize = 16;
    int arraySizeBytes = sizeof(int) * arraySize;
    int inputArray[] = {0, 1, 1, 2, 3, 5, 8, 13, 21, 34, 65, 99, 164, 263, 427, 690};

    int n_grids = 4;

    BlockParams blockParams = {
            arraySize / n_grids, 1, 1
    };

    GridParams gridParams = {
            n_grids, 1, 1
    };

    UniqueIndexCalculation program = UniqueIndexCalculation(blockParams, gridParams);
    program.run(inputArray, arraySize, arraySizeBytes);
}

int main() {
    Demo();
}
