#include "hip/hip_runtime.h"


#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <iostream>
#include <cmath>


__global__ void fibonacci_kernel(double* a, int n) {
    unsigned int index = threadIdx.x;
    if (index < n)
        a[index] = (pow((1 + sqrt(5.0)) / 2, index) - pow((1 - sqrt(5.0)) / 2, index)) / sqrt(5.0);
}


class Fibonacci {

public:

    int arrayLength;

    explicit Fibonacci(int arrayLength);

    void run(int numGrids, int numThreads) const;
    void displayResult(double* array, double* resultArray) const;
};

Fibonacci::Fibonacci(int arrayLength) {
    this->arrayLength = arrayLength;
}

void Fibonacci::displayResult(double *array, double* resultArray) const {
    for(int i = 0; i < this->arrayLength; i++)
        printf("Index %d: %f\n", i, resultArray[i]);
}

void Fibonacci::run(int numGrids, int numThreads) const {

    int deviceId = hipGetDevice(&deviceId);

    printf("GPU Device ID: %d\n", deviceId);
    printf("CPU Device ID: %d\n\n", hipCpuDeviceId);

    double * hostArray, * resultArray, * deviceArray;
    size_t arrayBytes = sizeof(int) * this->arrayLength;

    hipHostMalloc(&hostArray, arrayBytes, hipHostMallocDefault);
    hipHostMalloc(&resultArray, arrayBytes, hipHostMallocDefault);
    hipMalloc(&deviceArray, arrayBytes);

    hipMemcpy(deviceArray, hostArray, arrayBytes, hipMemcpyHostToDevice);

    fibonacci_kernel<<<numGrids, numThreads>>>(deviceArray, arrayLength);
    hipDeviceSynchronize();

    hipMemcpy(resultArray, deviceArray, arrayBytes, hipMemcpyDeviceToHost);

    displayResult(hostArray, resultArray);

    hipHostFree(hostArray);
    hipHostFree(resultArray);
    hipFree(deviceArray);
}


int main() {
    Fibonacci program(16);
    program.run(1, 256);
}
